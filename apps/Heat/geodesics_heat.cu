#include "hip/hip_runtime.h"
#include "rxmesh/attribute.h"
#include "rxmesh/matrix/dense_matrix.cuh"
#include "rxmesh/matrix/sparse_matrix.cuh"
#include "rxmesh/rxmesh_static.h"

using namespace rxmesh;

// 1st part - heat method
// setup LC & A matrix
// solve using chol like the template implementation to get u


template <typename T, uint32_t blockThreads>
__global__ static void setup_LC_matrix(const Context            context,
                                      const VertexAttribute<T>  coords,
                                      DenseMatrix<T>            LC_Mat,
                                      const bool use_uniform_laplace)
{
    // kernel for cotangent matrix setup
}


template <typename T, uint32_t blockThreads>
__global__ static void setup_A_matrix(const Context            context,
                                      const VertexAttribute<T> coords,
                                      SparseMatrix<T>          A_mat,
                                      const bool use_uniform_laplace,
                                      const T    time_step)
{
    // kernel for Area matrux setup
}

template <typename T>
void solve_using_chol(rxmesh::RXMeshStatic& rx)
{
    constexpr uint32_t blockThreads = 256;
    uint32_t num_vertices = rx.get_num_vertices();

    auto coords = rx.get_input_vertex_coordinates();

    SparseMatrix<T> A_mat(rx);                    
    DenseMatrix<T>  LC_Mat(rx, num_vertices, 3);  

    LaunchBox<blockThreads> launch_box_A;
    rx.prepare_launch_box({Op::VV},
                          launch_box_A,
                          (void*)setup_A_matrix<T, blockThreads>,
                          !Arg.use_uniform_laplace);
    setup_A_matrix<T, blockThreads>
        <<<launch_box_A.blocks,
           launch_box_A.num_threads,
           launch_box_A.smem_bytes_dyn>>>(rx.get_context(),
                                          *coords,
                                          A_mat,
                                          Arg.use_uniform_laplace,
                                          Arg.time_step);
    CUDA_ERROR(hipDeviceSynchronize());

    LaunchBox<blockThreads> launch_box_LC;
    rx.prepare_launch_box({Op::VV},
                          launch_box_LC,
                          (void*)setup_LC_matrix<T, blockThreads>,
                          !Arg.use_uniform_laplace);
    setup_LC_matrix<T, blockThreads><<<launch_box_LC.blocks,
                                       launch_box_LC.num_threads,
                                       launch_box_LC.smem_bytes_dyn>>>(
        rx.get_context(), *coords, LC_Mat, Arg.use_uniform_laplace);
    CUDA_ERROR(hipDeviceSynchronize());

    A_mat.pre_solve(PermuteMethod::NSTDIS);
    A_mat.solve(LC_Mat, *coords, Solver::CHOL);

    coords->move(rxmesh::DEVICE, rxmesh::HOST);

    rx.get_polyscope_mesh()->updateVertexPositions(*coords);
    polyscope::show();

    LC_Mat.release();
    A_mat.release();
}
